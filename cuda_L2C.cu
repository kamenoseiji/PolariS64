//	cuda_fft_xspec.c : FFT using CuFFT
//
//	Author : Seiji Kameno
//	Created: 2012/12/6
//
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string.h>
#include <math.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <errno.h>
// #include </usr/local/cuda/samples/common/inc/timer.h>
// #include <hip/hip_runtime.h>
// #include <timer.h>
#include "cuda_polaris.inc"
#define SCALEFACT 1.0/(NFFT* NsegSec)

int main(
	int		argc,			// Number of Arguments
	char	**argv )		// Pointer to Arguments
{
    int     rv;                     // Return value from OCTAVIA2
    unsigned char *vdifhead_ptr;    // VDIF header
    unsigned char *vdifdata_ptr;    // VDIF data
    unsigned char buf[VDIF_SIZE];   // 1312 bytes
    struct sockaddr_in  addr_recv, addr_send;   // Socked address
    struct ip_mreq  mreq;           // Multicast request
    int     sock_recv, sock_send;   // Receive and Send Socket ID
    int     frameID, threadID;      // frame and thread ID
    int     MaxFrameIndex = 199999;
    int     index;
//------------------------------------------ Open sockets
    sock_recv = socket(AF_INET, SOCK_DGRAM, 0);
    if(sock_recv < 0){
        perror("Socket Failed\n"); printf("%d\n", errno);
        return(-1);
    }
    addr_recv.sin_family = AF_INET;
    addr_recv.sin_port   = htons(60000);
    addr_recv.sin_addr.s_addr    = INADDR_ANY;
    if( bind(sock_recv, (struct sockaddr *)&addr_recv, sizeof(addr_recv)) < 0){
        perror("Bind Failed\n"); printf("%d\n", errno);
    }
    memset(buf, 0, sizeof(buf));
//------------------------------------------ Receive VDIF
    frameID = 0;
    while( frameID < MaxFrameIndex ){
        rv = recv(sock_recv, buf, sizeof(buf), 0);
        frameID    = (buf[5] << 16) + (buf[6] << 8) + buf[7];
    }
    threadID    = ((buf[12] & 0x03) << 8 ) + buf[13] - 1;
    printf("frameID = %06d threadID = %d\n", frameID, threadID);
    while(threadID < 1){
        rv = recv(sock_recv, buf, sizeof(buf), 0);
        frameID    = (buf[5] << 16) + (buf[6] << 8) + buf[7];
        threadID    = ((buf[12] & 0x03) << 8 ) + buf[13] - 1;
        printf("frameID = %06d : threadID = %d\n", frameID, threadID);
    }
//------------------------------------------ Repeat
    //while(1){
    for(index=0; index<1024; index++){
        rv = recv(sock_recv, buf, sizeof(buf), 0);
        frameID    = (buf[5] << 16) + (buf[6] << 8) + buf[7];
        threadID   = ((buf[12] & 0x03) << 8 ) + buf[13] - 1;
        printf("frame%d  thread%d \r", frameID, threadID);
    }
    close(sock_recv);
}
