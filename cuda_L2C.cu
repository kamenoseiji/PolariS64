//	cuda_fft_xspec.c : FFT using CuFFT
//
//	Author : Seiji Kameno
//	Created: 2012/12/6
//
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string.h>
#include <math.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <errno.h>
// #include </usr/local/cuda/samples/common/inc/timer.h>
// #include <hip/hip_runtime.h>
// #include <timer.h>
#include "cuda_polaris.inc"
#define SCALEFACT 1.0/(NFFT* NsegSec)

int main(
	int		argc,			// Number of Arguments
	char	**argv )		// Pointer to Arguments
{
    int     rv, sd;                 // Return value from OCTAVIA2
    unsigned char *vdifhead_ptr;    // VDIF header
    unsigned char *vdifdata_ptr;    // VDIF data
    unsigned char buf[VDIF_SIZE];   // 1312 bytes
    struct sockaddr_in  addr_recv, addr_send;   // Socked address
    struct ip_mreq  mreq;           // Multicast request
    int     sock_recv, sock_send;   // Receive and Send Socket ID
    int     frameID, threadID;      // frame and thread ID
    int     MaxFrameIndex = 199999;
    int     index;
//------------------------------------------ Open sockets
    sock_recv = socket(AF_INET, SOCK_DGRAM, 0);
    sock_send = socket(AF_INET, SOCK_DGRAM, 0);
    if(sock_recv < 0){
        perror("Socket (recv) Failed\n"); printf("%d\n", errno);
        return(-1);
    }
    if(sock_send < 0){
        perror("Socket (send) Failed\n"); printf("%d\n", errno);
        return(-1);
    }
    addr_recv.sin_family = AF_INET;
    addr_recv.sin_port   = htons(60000);
    addr_recv.sin_addr.s_addr    = INADDR_ANY;
    if( bind(sock_recv, (struct sockaddr *)&addr_recv, sizeof(addr_recv)) < 0){
        perror("Bind Failed (recv)\n"); printf("%d\n", errno);
    }
    addr_send.sin_family = AF_INET;
    addr_send.sin_port   = htons(60000);
    addr_send.sin_addr.s_addr    = htonl(INADDR_ANY);
    if( bind(sock_send, (struct sockaddr *)&addr_send, sizeof(addr_send)) < 0){
        perror("Bind Failed (send)\n"); printf("%d\n", errno);
    }
    memset(buf, 0, sizeof(buf));
//------------------------------------------ Receive VDIF
    frameID = 0;
    while( frameID < MaxFrameIndex ){
        rv = recv(sock_recv, buf, sizeof(buf), 0);
        frameID    = (buf[5] << 16) + (buf[6] << 8) + buf[7];
    }
    threadID    = ((buf[12] & 0x03) << 8 ) + buf[13] - 1;
    printf("frameID = %06d threadID = %d\n", frameID, threadID);
    while(threadID < 1){
        rv = recv(sock_recv, buf, sizeof(buf), 0);
        frameID    = (buf[5] << 16) + (buf[6] << 8) + buf[7];
        threadID    = ((buf[12] & 0x03) << 8 ) + buf[13] - 1;
        printf("frameID = %06d : threadID = %d\n", frameID, threadID);
    }
//------------------------------------------ Repeat
    printf("HIDOI\n");
    //while(1){
    frameID = 0;
    while(frameID < MaxFrameIndex){
        rv = recv(sock_recv, buf, sizeof(buf), 0);
        frameID    = (buf[5] << 16) + (buf[6] << 8) + buf[7];
        threadID   = ((buf[12] & 0x03) << 8 ) + buf[13] - 1;
        printf("frame%d  thread%d \r", frameID, threadID);
        sd = send(sock_send, buf, sizeof(buf), 0);
    }
    close(sock_recv);
    close(sock_send);
}
