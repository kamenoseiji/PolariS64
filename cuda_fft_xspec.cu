#include "hip/hip_runtime.h"
//	cuda_fft_xspec.c : FFT using CuFFT
//
//	Author : Seiji Kameno
//	Created: 2012/12/6
//
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string.h>
#include <math.h>
// #include </usr/local/cuda/samples/common/inc/timer.h>
// #include <hip/hip_runtime.h>
// #include <timer.h>
#include "cuda_polaris.inc"
#define SCALEFACT 1.0/(NFFT* NsegSec)

int	segment_offset(struct SHM_PARAM	*, int *);
int	fileRecOpen(struct SHM_PARAM	*, char *, FILE **, FILE **, FILE **);

main(
	int		argc,			// Number of Arguments
	char	**argv )		// Pointer to Arguments
{
	int		shrd_param_id;				// Shared Memory ID
	int		index;						// General Index
	int		threadID;					// Index for thread (= IF stream)
	int		seg_index;					// Index for Segment
	int		offset[16384];				// Segment offset position
    int     PageSize;                   // Page size [bytes]
	struct	SHM_PARAM	*param_ptr;		// Pointer to the Shared Param
	struct	sembuf		sops;			// Semaphore for data access
	unsigned char	*vdifdata_ptr;		// Pointer to shared VDIF data
	float	*xspec_ptr;					// Pointer to 1-sec-integrated Power Spectrum
	FILE	*Pfile_ptr[16];				// Power-Meter File Pointer to write
	FILE	*Afile_ptr[16];				// Autocorr File Pointer to write
	FILE	*Cfile_ptr[16];				// Cross corr File Pointer to write
	char	fname_pre[16];

	//-------- CUDA data
	dim3			Dg, Db(512,1, 1);	// Grid and Block size
	unsigned char	*cuvdifdata_ptr;	// Pointer to VDIF data in GPU
	hipfftHandle		cufft_plan;			// 1-D FFT Plan, to be used in cufft
	hipfftReal		*cuRealData;		// Time-beased data before FFT, every IF, every segment
	hipfftComplex	*cuSpecData;		// FFTed spectrum, every IF, every segment
	float			*cuPowerSpec;		// (autocorrelation) Power Spectrum
	float2			*cuXSpec;           // cross power spectrum
    hipEvent_t     start, stop;        // Time-mesurement events
	int				modeSW = 0;
    float           elapsed_time_ms;

	//-------- Pointer to functions
 	void	(*segform[4])( unsigned char *, float *, int);
 	segform[0] = segform_1bit;
 	segform[1] = segform_2bit;
 	segform[2] = segform_4bit;
 	segform[3] = segform_8bit;
    hipEventCreate(&start);
    hipEventCreate(&stop);
//------------------------------------------ Access to the SHARED MEMORY
	shrd_param_id = shmget( SHM_PARAM_KEY, sizeof(struct SHM_PARAM), 0444);
	param_ptr  = (struct SHM_PARAM *)shmat(shrd_param_id, NULL, 0);
	vdifdata_ptr = (unsigned char *)shmat(param_ptr->shrd_vdifdata_id, NULL, SHM_RDONLY);
	xspec_ptr  = (float *)shmat(param_ptr->shrd_xspec_id, NULL, 0);
    PageSize = param_ptr->fsample  / 8 / PAGEPERSEC * param_ptr->qbit;
	switch( param_ptr->qbit ){
 		case  1 :	modeSW = 0; break;
 		case  2 :	modeSW = 1; break;
 		case  4 :	modeSW = 2; break;
 		case  8 :	modeSW = 3; break;
 	}
//------------------------------------------ Prepare for CuFFT
	hipMalloc( (void **)&cuvdifdata_ptr, PageSize);                                    // for Sampled Data
	hipMalloc( (void **)&cuRealData, NsegPage* NFFT* sizeof(hipfftReal) );              // For FFT segments in a page
	hipMalloc( (void **)&cuSpecData, NST* NsegPage* NFFTC* sizeof(hipfftComplex) );     // For FFTed spectra
	hipMalloc( (void **)&cuPowerSpec,NST* NFFT2* sizeof(float));                       // For autcorr spectra
	hipMalloc( (void **)&cuXSpec,    NST* NFFT2* sizeof(float2)/ 2);                   // For cross-corr spectra
	if(hipGetLastError() != hipSuccess){
	 	fprintf(stderr, "Cuda Error : Failed to allocate memory.\n"); return(-1); }

 	if(hipfftPlan1d(&cufft_plan, NFFT, HIPFFT_R2C, NsegPage ) != HIPFFT_SUCCESS){
 		fprintf(stderr, "Cuda Error : Failed to create plan.\n"); return(-1); }
//------------------------------------------ Parameters for S-part format
 	segment_offset(param_ptr, offset);
	// for(seg_index=0; seg_index< NsegPage; seg_index++){	printf("Offset[%d] = %d\n", seg_index, offset[seg_index]);}
//------------------------------------------ K5 Header and Data
	hipMemset( cuPowerSpec, 0, NST* NFFT2* sizeof(float));		// Clear Power Spectrum to accumulate
 	param_ptr->current_rec = -1;
	setvbuf(stdout, (char *)NULL, _IONBF, 0);   // Disable stdout cache
	while(param_ptr->validity & ACTIVE){
		if( param_ptr->validity & (FINISH + ABSFIN) ){  break; }

		//-------- Initial setup for cycles
		hipMemset( cuPowerSpec, 0, NST* NFFT2* sizeof(float));		// Clear Power Spectrum to accumulate
		hipMemset( cuXSpec, 0, NST* NFFT2* sizeof(float2)/2);		// Clear Power Spectrum to accumulate

		//-------- Open output files
		if(param_ptr->current_rec == 0){
			sprintf(fname_pre, "%04d%03d%02d%02d%02d", param_ptr->year, param_ptr->doy, param_ptr->hour, param_ptr->min, param_ptr->sec );
			fileRecOpen(param_ptr, fname_pre, Pfile_ptr, Afile_ptr, Cfile_ptr);
		}
		//-------- Wait for S-part memory 
		sops.sem_num = (ushort)SEM_VDIF_PART; sops.sem_op = (short)-1; sops.sem_flg = (short)0;
		semop( param_ptr->sem_data_id, &sops, 1);
		usleep(8);	// Wait 0.01 msec
		// StartTimer();
        hipEventRecord(start, 0);
        for(threadID=0; threadID < NST; threadID++){
		    //-------- SHM -> GPU memory transfer
		    hipMemcpy(cuvdifdata_ptr, &vdifdata_ptr[PageSize* (threadID*2 + param_ptr->part_index)], PageSize, hipMemcpyHostToDevice);
		    //-------- Segment Format
		    Dg.x=NFFT/512; Dg.y=1; Dg.z=1;
		    for(index=0; index < NsegPage; index ++){
			    (*segform[modeSW])<<<Dg, Db>>>( &cuvdifdata_ptr[offset[index]], &cuRealData[index* NFFT], NFFT);
		    }

		    //-------- FFT Real -> Complex spectrum
		    hipDeviceSynchronize();
		    hipfftExecR2C(cufft_plan, cuRealData, cuSpecData);		// FFT Time -> Freq
		    hipDeviceSynchronize();

		    //---- Auto Corr
		    Dg.x= NFFT/512; Dg.y=1; Dg.z=1;
		    for(seg_index=0; seg_index<NsegPage; seg_index++){
				accumPowerSpec<<<Dg, Db>>>( &cuSpecData[seg_index* NFFTC], &cuPowerSpec[threadID* NFFT2],  NFFT2);
			}
		}
		//---- Cross Corr
		for(seg_index=0; seg_index<NsegPage; seg_index++){
		    // accumCrossSpec<<<Dg, Db>>>( &cuSpecData[seg_index* NFFTC], &cuSpecData[(seg_index + NsegPage)* NFFTC], cuXSpec,  NFFT2);
		    accumCrossSpec<<<Dg, Db>>>( &cuSpecData[seg_index* NFFTC], &cuSpecData[seg_index* NFFTC], cuXSpec,  NFFT2);
		}
		// printf("%lf [msec]\n", GetTimer());
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time_ms, start, stop);
		printf("%4d %03d %02d:%02d:%02d %8.2f [msec]\n", param_ptr->year, param_ptr->doy, param_ptr->hour, param_ptr->min, param_ptr->sec, elapsed_time_ms);

		//-------- Dump cross spectra to shared memory
		// if( param_ptr->buf_index == PARTNUM - 1){
		hipMemcpy(xspec_ptr, cuPowerSpec, NST* NFFT2* sizeof(float), hipMemcpyDeviceToHost);
		sops.sem_num = (ushort)SEM_FX; sops.sem_op = (short)1; sops.sem_flg = (short)0; semop( param_ptr->sem_data_id, &sops, 1);
		for(index=0; index<NST; index++){
		    if(Afile_ptr[index] != NULL){fwrite(&xspec_ptr[index* NFFT2], sizeof(float), NFFT2, Afile_ptr[index]);}   // Save Power Spectra
			if(Pfile_ptr[index] != NULL){fwrite(&(param_ptr->power[index]), sizeof(float), 1, Pfile_ptr[index]);}   // Save Power
		}
		hipMemcpy(&xspec_ptr[NST* NFFT2], cuXSpec, NFFT2* sizeof(float2), hipMemcpyDeviceToHost);
		if(Cfile_ptr[0] != NULL){fwrite(&xspec_ptr[NST* NFFT2], sizeof(float2), NFFT2, Cfile_ptr[0]);}   // Save Cross Spectra

	    //-------- Refresh output data file
		if(param_ptr->current_rec == MAX_FILE_REC - 1){
			for(index=0; index<param_ptr->num_st; index++){
				if( Afile_ptr[index] != NULL){   fclose(Afile_ptr[index]);}
				if( Pfile_ptr[index] != NULL){   fclose(Pfile_ptr[index]);}
				if( Cfile_ptr[0] != NULL){   fclose(Cfile_ptr[0]);}
			}
			param_ptr->current_rec = 0;
		} else { param_ptr->current_rec ++;}
		// param_ptr->current_rec ++;
	}	// End of part loop
/*
-------------------------------------------- RELEASE the SHM
*/
	for(index=0; index<param_ptr->num_st; index++){
		if( Afile_ptr[index] != NULL){	fclose(Afile_ptr[index]);}
		if( Pfile_ptr[index] != NULL){	fclose(Pfile_ptr[index]);}
		if( Cfile_ptr[0] != NULL){	fclose(Cfile_ptr[0]);}
	}
	hipfftDestroy(cufft_plan);
	hipFree(cuvdifdata_ptr); hipFree(cuRealData); hipFree(cuSpecData); hipFree(cuPowerSpec); hipFree(cuXSpec);

    return(0);
}

//-------- Offset to the pointer of  segmant
int	segment_offset(
	struct SHM_PARAM	*param_ptr,	// Pointer to shared parameter
	int					*offset_ptr)
{
	int			seg_index;		// Index for segments
	long long	SegLenByte;		// Length of a segment in Bytes
	SegLenByte = param_ptr->segLen / 8 * param_ptr->qbit;		// Segment Length in Byte
	for(seg_index = 0; seg_index < param_ptr->segPage; seg_index ++){
		offset_ptr[seg_index]= SegLenByte* seg_index;
    }
	return(param_ptr->segPage);
}

//-------- Open Files to Record Data
int	fileRecOpen(
	struct SHM_PARAM	*param_ptr,		// IN: Shared Parameter
	char				*fname_pre,		// IN: File name prefix
	FILE				**Pfile_ptr,	//OUT: file pointer
	FILE				**Afile_ptr,	//OUT: file pointer
	FILE				**Cfile_ptr)	//OUT: file pointer
{
	char				fname[24];
	int					file_index;		// IN: File index number

	for(file_index=0; file_index < param_ptr->num_st; file_index++){
		if( param_ptr->AC_REC & (P00_REC << file_index) ){		// P file
			sprintf(fname, "%s.%c.%02d", fname_pre, 'P', file_index);
			Pfile_ptr[file_index] = fopen(fname, "w");
			fwrite( param_ptr, sizeof(struct SHM_PARAM), 1, Pfile_ptr[file_index]);
		} else { Pfile_ptr[file_index] = NULL;}

		if( param_ptr->AC_REC & (A00_REC << file_index) ){		// A file
			sprintf(fname, "%s.%c.%02d", fname_pre, 'A', file_index);
			Afile_ptr[file_index] = fopen(fname, "w");
			fwrite( param_ptr, sizeof(struct SHM_PARAM), 1, Afile_ptr[file_index]);
		} else { Afile_ptr[file_index] = NULL;}
	}
    if( param_ptr->XC_REC & 0x01){		// C file
        sprintf(fname, "%s.%c.%02d", fname_pre, 'C', 0);
        Cfile_ptr[0] = fopen(fname, "w");
        fwrite( param_ptr, sizeof(struct SHM_PARAM), 1, Cfile_ptr[0]);
    } else { Cfile_ptr[0] = NULL;}
	return(0);
}
